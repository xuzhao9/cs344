#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Grean and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "utils.h"

__device__
int next_power_of_2(int sz) {
}

__device__
int _pixel_coordinate_to_offset(int x, int y, int numRows, int numCols) {

}

__device__
void _transform(const uchar4* const rgbaImage,
                unsigned char* const greyImage,
                int pixel_offset) {
  greyImage[pixel_offset] = rgbaImage[pixel_offset].R * 0.299f + \
                            rgbaImage[pixel_offset].G * 0.587f + \
                            rgbaImage[pixel_offset].B * 0.114f;
}

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  //TODO
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion

  //First create a mapping from the 2D block and grid locations
  //to an absolute 2D location in the image, then use that to
  //calculate a 1D offset

  //Suppose the image format is row-major
  // get number of pixels per threads
  numRowsRounded = next_power_of_2(numRows);
  numColsRounded = next_power_of_2(numCols);
  rowSize = numRowsRounded / (blockDim.x * gridDim.x);
  colSize = numColsRounded / (blockDim.y * gridDim.y);
  //get the top-left location id
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  //calculate the top-left coordinate
  int top_left_pixel_x = numCols * idx;
  int top_left_pixel_y = colSize * idy;
  //traverse each pixel
  for (int i = 0; i < rowSize; i ++) {
    for (int j = 0; j < colSize; j ++) {
      int pixelx = top_left_pixel_x + i;
      int pixely = top_left_pixel_y + i;
      int pixel_offset = _pixel_coordinate_to_offset(pixelx, pixely, numRows, numCols);
      if (pixelx < numRows && pixely < numCols) {
        _transform(rgbaImage, greyImage, pixel_offset);
      }
    }
  }
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  //You must fill in the correct sizes for the blockSize and gridSize
  //currently only one block with one thread is being launched
  const dim3 blockSize(1, 1, 1);  //TODO
  const dim3 gridSize( 1, 1, 1);  //TODO
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}
